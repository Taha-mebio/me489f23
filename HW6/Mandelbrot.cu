#include "hip/hip_runtime.h"
%%cu
/*******************************************************************************
To compile: gcc -O3 -o mandelbrot mandelbrot.c -lm
To create an image with 4096 x 4096 pixels: ./mandelbrot 4096 4096
*******************************************************************************/
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"

int writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI);

#define MXITER 1000
#define BLOCKSIZE 1024


/*******************************************************************************/
// Define a complex number
typedef struct {
    double x;
    double y;
} complex_t;

/*******************************************************************************/
// Return iterations before z leaves mandelbrot set for given c
__device__ int testpoint(complex_t c){
  int iter;
  complex_t z = c;

  for(iter=0; iter<MXITER; iter++){
    // real part of z^2 + c
    double tmp = (z.x*z.x) - (z.y*z.y) + c.x;
    // update with imaginary part of z^2 + c
    z.y = z.x*z.y*2. + c.y;
    // update real part
    z.x = tmp;
    // check bound
    if((z.x*z.x+z.y*z.y)>4.0){ return iter;}
  }
  return iter;
}

/*******************************************************************************/
// perform Mandelbrot iteration on a grid of numbers in the complex plane
// record the  iteration counts in the count array
__global__ void mandelKernel(int Nre, int Nim, complex_t *d_cmin, complex_t *d_dc, float* d_count){

// Create an integer for the x and y coordinates of the pixel/thread
  int m = blockIdx.x * blockDim.x + threadIdx.x;
  int n = blockIdx.y * blockDim.y + threadIdx.y;

  if(m<Nre && n<Nim){
    complex_t c;
    c.x = d_cmin->x + d_dc->x*m;
    c.y = d_cmin->y + d_dc->y*n;
    d_count[m+n*Nre] = (float) testpoint(c);
    }
}

/*******************************************************************************/
int main(int argc, char **argv){
  hipError_t cudaStatus;

  // to create a 4096x4096 pixel image
  // usage: ./mandelbrot 4096 4096

  int p_w = 32;

  // number of pixels in the real/horizantal direction.
  int Nre = (argc==3) ? atoi(argv[1]): 8192;
  // number of pixels in the imaginary/vertical direction.
  int Nim = (argc==3) ? atoi(argv[2]): 8192;

  // Parameters for a bounding box for "c" that generates an interesting image
  // const float centRe = -.759856, centIm= .125547;
  // const float diam  = 0.151579;
  const float centRe = -0.5, centIm= 0;
  const float diam  = 3.0;

  complex_t cmin;
  complex_t cmax;
  complex_t dc;

  cmin.x = centRe - 0.5*diam;
  cmax.x = centRe + 0.5*diam;
  cmin.y = centIm - 0.5*diam;
  cmax.y = centIm + 0.5*diam;

  //set step sizes
  dc.x = (cmax.x-cmin.x)/(Nre-1);
  dc.y = (cmax.y-cmin.y)/(Nim-1);

  float *count;
  count = (float*) malloc(Nre*Nim*sizeof(float));

  // ON DEVICE
  float *d_count;
  hipMalloc((void**)&d_count,Nre*Nim*sizeof(float));

  // Allocate memory for the complex_t struct on the GPU
  complex_t *d_cmin;
  complex_t *d_dc;

  hipMalloc((void **)&d_cmin, sizeof(complex_t));
  hipMalloc((void **)&d_dc, sizeof(complex_t));


  // Copy the struct data from host to device
  hipMemcpy(d_cmin, &cmin, sizeof(complex_t), hipMemcpyHostToDevice);
  hipMemcpy(d_dc, &dc, sizeof(complex_t), hipMemcpyHostToDevice);
  hipMemcpy(d_count, count, Nre * Nim * sizeof(float), hipMemcpyHostToDevice);

  dim3 block_dim(p_w, p_w, 1);
  dim3 grid_dim(((Nre + p_w - 1 )/p_w),((Nim + p_w - 1 )/p_w), 1);

  //start time in CPU cycles
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // compute mandelbrot set
  mandelKernel<<<grid_dim, block_dim>>>(Nre, Nim, d_cmin, d_dc, d_count);

  hipMemcpy(count, d_count, Nre * Nim * sizeof(float), hipMemcpyDeviceToHost);

  // copy from the GPU back to the host here
  //start time in CPU cycles
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // print elapsed time
  printf("elapsed = %f seconds\n", (milliseconds/1000));

  // output mandelbrot to ppm format image
  printf("Printing mandelbrot.ppm...");
  writeMandelbrot("mandelbrot.ppm", Nre, Nim, count, 0, 80);
  printf("done.\n");
  hipEventDestroy(start);
  hipEventDestroy(stop);
  free(count);
  hipFree(d_count);
  hipFree(d_cmin);
  hipFree(d_dc);

  exit(0);
  return 0;
}


/* Output data as PPM file */
void saveppm(const char *filename, unsigned char *img, int width, int height){

  /* FILE pointer */
  FILE *f;

  /* Open file for writing */
  f = fopen(filename, "wb");

  /* PPM header info, including the size of the image */
  fprintf(f, "P6 %d %d %d\n", width, height, 255);

  /* Write the image data to the file - remember 3 byte per pixel */
  fwrite(img, 3, width*height, f);

  /* Make sure you close the file */
  fclose(f);
}



int writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI){

  int n, m;
  unsigned char *rgb   = (unsigned char*) calloc(3*width*height, sizeof(unsigned char));

  for(n=0;n<height;++n){
    for(m=0;m<width;++m){
      int id = m+n*width;
      int I = (int) (768*sqrt((double)(img[id]-minI)/(maxI-minI)));

      // change this to change palette
      if(I<256)      rgb[3*id+2] = 255-I;
      else if(I<512) rgb[3*id+1] = 511-I;
      else if(I<768) rgb[3*id+0] = 767-I;
      else if(I<1024) rgb[3*id+0] = 1023-I;
      else if(I<1536) rgb[3*id+1] = 1535-I;
      else if(I<2048) rgb[3*id+2] = 2047-I;

    }
  }

  saveppm(fileName, rgb, width, height);

  free(rgb);
}
